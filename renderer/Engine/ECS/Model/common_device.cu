#include "hip/hip_runtime.h"
//
// Created by lenovo on 10/14/2024.
//

#include "common_device.cuh"

__global__  void transformVerticesKernel(float3* vertices, float3 center, uint32_t numVertices){
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numVertices) {
        vertices[idx].x -= center.x;
        vertices[idx].y -= center.y;
        vertices[idx].z -= center.z;
    }
}


void gpu::transformVerticesCUDA(float3 *vertices, float3 center, uint32_t numVertices) {
    uint32_t threadsPerBlock = 256;
    uint32_t blocksPerGrid = (numVertices + threadsPerBlock - 1) / threadsPerBlock;
    transformVerticesKernel<<<blocksPerGrid, threadsPerBlock>>>(vertices, center, numVertices);
    hipDeviceSynchronize();
}

